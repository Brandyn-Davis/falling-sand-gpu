#include "hip/hip_runtime.h"
//#include "gputk.h"
#include "sand_simulator_gpu_update.h"

static const char BLK_SIZE = 32;
static const char EMPTY = '.';
static const char SAND = '*';
static const int COARSE = 2;

__global__ void updateKernel(
    const char* privateGrid,
    char* grid,
    int height,
    int width) {

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int idx = y * width + x;

        if (privateGrid[idx] == SAND) {  // Check the previous state from privateGrid
            // Down
            if (y + 1 < height && privateGrid[(y + 1) * width + x] == EMPTY) {
                grid[(y + 1) * width + x] = SAND;
                grid[idx] = EMPTY;
            }
            // Down-left or Down-right
            else if (y + 1 < height) {
                if (x - 1 >= 0 && privateGrid[(y + 1) * width + x - 1] == EMPTY) {
                    // Move down-left
                    grid[idx] = EMPTY;
                    grid[(y + 1) * width + x - 1] = SAND;
                }
                else if (x + 1 < width && privateGrid[(y + 1) * width + x + 1] == EMPTY) {
                    // Move down-right
                    grid[idx] = EMPTY;
                    grid[(y + 1) * width + x + 1] = SAND;
                }
            }
        }
    }
}

void updateSand(const char* privateGrid, char* grid, int height, int width) {
    // Allocate device memory
    char *d_privateGrid;
    char *d_grid;
    gpuErrchk( hipMalloc((void**)&d_privateGrid, height * width) );
    gpuErrchk( hipMalloc((void**)&d_grid, height * width) );

    // Copy to device
    gpuErrchk( hipMemcpy((void*)d_privateGrid, (void*)privateGrid, height * width, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpy((void*)d_grid, (void*)grid, height * width, hipMemcpyHostToDevice) );

    // Define launch parameters
    dim3 _blockDim(BLK_SIZE, BLK_SIZE);
    dim3 _gridDim((width + BLK_SIZE * COARSE - 1) / BLK_SIZE * COARSE,
                  (height + BLK_SIZE * COARSE - 1) / BLK_SIZE * COARSE);

    // Run kernel
    updateKernel<<<_gridDim, _blockDim>>>(d_privateGrid, d_grid, height, width);
    gpuErrchk( hipDeviceSynchronize() );

    // Copy result back to host
    gpuErrchk( hipMemcpy((void*)grid, (void*)d_grid, height * width, hipMemcpyDeviceToHost) );

    // Free device memory
    gpuErrchk( hipFree(d_privateGrid) );
    gpuErrchk( hipFree(d_grid) );
}
